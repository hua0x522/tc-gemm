#include "hip/hip_runtime.h"
#include "utils.h"
#include "base.cuh"

#define MMA_M 16
#define MMA_N 8
#define MMA_K 16

#define WARP_SIZE 32

__global__ void mma_base_kernel(int M, int N, int K, half* d_A, half* d_B, half* d_C) {
    const int k_tiles = ROUND(K, MMA_K);
    const int warp_row = blockIdx.x * MMA_M;
    const int warp_col = blockIdx.y * MMA_N;
    const int lane_id = threadIdx.x % WARP_SIZE;

    __shared__ half A_smem[MMA_M][MMA_K];
    __shared__ half B_smem[MMA_N][MMA_K];
    __shared__ half C_smem[MMA_M][MMA_N];

    uint32_t RA[4];
    uint32_t RB[2];
    uint32_t RC[2] = {0, 0};

    for (size_t i = 0; i < k_tiles; i++) {
        *((int4*)(&A_smem[lane_id / 2][0]) + lane_id % 2) = 
            *((int4*)(&d_A[(warp_row + lane_id / 2) * K + i * MMA_K]) + lane_id % 2);

        if (lane_id < MMA_N * 2) {
            *((int4*)(&B_smem[lane_id / 2][0]) + lane_id % 2) = 
                *((int4*)(&d_B[i * MMA_K + (warp_col + lane_id / 2) * K]) + lane_id % 2);
        }

        __syncthreads();
    
        uint32_t A_smem_lane_addr = __cvta_generic_to_shared(&A_smem[lane_id % 16][(lane_id / 16) * 8]);
        LDMATRIX_X4(RA[0], RA[1], RA[2], RA[3], A_smem_lane_addr);

        uint32_t B_smem_lane_addr = __cvta_generic_to_shared(&B_smem[lane_id % 8][((lane_id / 8) % 2) * 8]);
        LDMATRIX_X2(RB[0], RB[1], B_smem_lane_addr);

        HMMA16816(RC[0], RC[1], RA[0], RA[1], RA[2], RA[3], RB[0], RB[1], RC[0], RC[1]);

        __syncthreads();
    }

    *((uint32_t *)(&C_smem[lane_id / 4][0]) + lane_id % 4) = RC[0];
    *((uint32_t *)(&C_smem[lane_id / 4 + 8][0]) + lane_id % 4) = RC[1];

    __syncthreads();

    if (lane_id < MMA_M) {
        *((int4 *)(&d_C[(warp_row + lane_id) * N + warp_col])) = *((int4 *)(&C_smem[lane_id][0]));
    }
}

void mma_base(int M, int N, int K, half* h_A, half* h_B, half* h_C) {
    transpose(h_B, K, N);

    half* d_A;
    half* d_B;
    half* d_C;
    struct timeval tv;
    double start, end;

    hipMalloc(&d_A, M * K * sizeof(half));
    hipMalloc(&d_B, K * N * sizeof(half));
    hipMalloc(&d_C, M * N * sizeof(half));
    hipMemcpy(d_A, h_A, M * K * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, M * N * sizeof(half), hipMemcpyHostToDevice);

    gettimeofday(&tv, nullptr);
    start = tv.tv_sec + tv.tv_usec / 1.0e6;

    mma_base_kernel<<<dim3(ROUND(M, 16), ROUND(N, 16)), dim3(16, 16)>>>(M, N, K, d_A, d_B, d_C);

    hipDeviceSynchronize();
    gettimeofday(&tv, nullptr);
    end = tv.tv_sec + tv.tv_usec / 1.0e6;
    printf("mma base time: %lf\n", end - start);

    hipMemcpy(h_C, d_C, M * N * sizeof(half), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}