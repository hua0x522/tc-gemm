#include "hip/hip_runtime.h"
#include "utils.h"

__global__ void matmul_kernel(int M, int N, int K, half* d_A, half* d_B, half* d_C) {
    int m = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;

    half sum = 0.0;
    for (int k = 0; k < K; k++) {
        sum += d_A[m * K + k] * d_B[k * N + n];
    }
    d_C[m * N + n] = sum;
}

void cuda_core(int M, int N, int K, half* h_A, half* h_B, half* h_C) {
    half* d_A;
    half* d_B;
    half* d_C;
    struct timeval tv;
    double start, end;

    hipMalloc(&d_A, M * K * sizeof(half));
    hipMalloc(&d_B, K * N * sizeof(half));
    hipMalloc(&d_C, M * N * sizeof(half));
    hipMemcpy(d_A, h_A, M * K * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, M * N * sizeof(half), hipMemcpyHostToDevice);

    gettimeofday(&tv, nullptr);
    start = tv.tv_sec + tv.tv_usec / 1.0e6;

    matmul_kernel<<<dim3(ROUND(M, 16), ROUND(N, 16)), dim3(16, 16)>>>(M, N, K, d_A, d_B, d_C);

    hipDeviceSynchronize();
    gettimeofday(&tv, nullptr);
    end = tv.tv_sec + tv.tv_usec / 1.0e6;
    printf("cuda core time: %lf\n", end - start);

    hipMemcpy(h_C, d_C, M * N * sizeof(half), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
